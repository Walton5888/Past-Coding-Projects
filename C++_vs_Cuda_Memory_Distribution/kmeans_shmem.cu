#include "hip/hip_runtime.h"
// kmeans_shmem.cu
#include "kmeans_kernel.cuh"
#include "kmeans_implementations.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/extrema.h>
#include <cmath>

// Shared memory optimized kernels

__global__ void findNearestCentroidSharedKernel(const float* points, const float* centroids, int* labels, int numPoints, int numCentroids, int dims) {
    extern __shared__ float sharedCentroids[];
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load centroids into shared memory
    for (int i = threadIdx.x; i < numCentroids * dims; i += blockDim.x) {
        sharedCentroids[i] = centroids[i];
    }
    __syncthreads();
    
    if (idx < numPoints) {
        float minDistance = INFINITY;
        int nearestCentroid = 0;
        for (int c = 0; c < numCentroids; c++) {
            float distance = 0;
            for (int d = 0; d < dims; d++) {
                float diff = points[idx * dims + d] - sharedCentroids[c * dims + d];
                distance += diff * diff;
            }
            if (distance < minDistance) {
                minDistance = distance;
                nearestCentroid = c;
            }
        }
        labels[idx] = nearestCentroid;
    }
}

// You can add more shared memory optimized functions here

KMeansResult runCUDAShmemKMeans(const std::vector<float>& points, const KMeansParams& params) {
    int numPoints = points.size() / params.dims;
    
    // Create device vectors
    thrust::device_vector<float> d_points = points;
    thrust::device_vector<float> d_centroids(params.k * params.dims);
    thrust::device_vector<float> d_newCentroids(params.k * params.dims);
    thrust::device_vector<int> d_labels(numPoints);
    thrust::device_vector<int> d_clusterSizes(params.k);
    thrust::device_vector<int> d_converged(1);

    // Initialize centroids
    int blockSize = 256;
    int gridSize = (params.k + blockSize - 1) / blockSize;
    initializeCentroidsKernel<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(d_centroids.data()),
        thrust::raw_pointer_cast(d_points.data()),
        numPoints, params.k, params.dims, params.seed
    );

    int iterations = 0;
    bool converged = false;
    while (iterations < params.maxIterations && !converged) {
        // Find nearest centroid for each point using shared memory
        gridSize = (numPoints + blockSize - 1) / blockSize;
        int sharedMemSize = params.k * params.dims * sizeof(float);
        findNearestCentroidSharedKernel<<<gridSize, blockSize, sharedMemSize>>>(
            thrust::raw_pointer_cast(d_points.data()),
            thrust::raw_pointer_cast(d_centroids.data()),
            thrust::raw_pointer_cast(d_labels.data()),
            numPoints, params.k, params.dims
        );

        // Reset new centroids and cluster sizes
        thrust::fill(d_newCentroids.begin(), d_newCentroids.end(), 0.0f);
        thrust::fill(d_clusterSizes.begin(), d_clusterSizes.end(), 0);

        // Update centroids
        updateCentroidsKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_points.data()),
            thrust::raw_pointer_cast(d_labels.data()),
            thrust::raw_pointer_cast(d_newCentroids.data()),
            thrust::raw_pointer_cast(d_clusterSizes.data()),
            numPoints, params.k, params.dims
        );

        // Normalize centroids
        gridSize = (params.k + blockSize - 1) / blockSize;
        normalizeCentroidsKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_newCentroids.data()),
            thrust::raw_pointer_cast(d_clusterSizes.data()),
            params.k, params.dims
        );

        // Check for convergence
        thrust::fill(d_converged.begin(), d_converged.end(), 1);
        checkConvergenceKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_centroids.data()),
            thrust::raw_pointer_cast(d_newCentroids.data()),
            params.k, params.dims, params.threshold,
            thrust::raw_pointer_cast(d_converged.data())
        );

        converged = d_converged[0] == 1;

        // Update centroids for next iteration
        d_centroids = d_newCentroids;

        iterations++;
    }

    // Copy results back to host
    thrust::host_vector<int> h_labels = d_labels;
    thrust::host_vector<float> h_centroids = d_centroids;

    std::vector<int> labels(h_labels.begin(), h_labels.end());
    std::vector<std::vector<float>> finalCentroids(params.k, std::vector<float>(params.dims));
    for (int i = 0; i < params.k; i++) {
        for (int d = 0; d < params.dims; d++) {
            finalCentroids[i][d] = h_centroids[i * params.dims + d];
        }
    }

    return KMeansResult{labels, finalCentroids, iterations, 0.0}; // Time per iteration is calculated in main.cpp
}