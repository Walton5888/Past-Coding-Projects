#include "hip/hip_runtime.h"
// kmeans_basic.cu
#include "kmeans_kernel.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/extrema.h>
#include <cmath>

__device__ float calculateDistanceBasic(const float* a, const float* b, int dims) {
    float sum = 0;
    for (int i = 0; i < dims; i++) {
        float diff = a[i] - b[i];
        sum += diff * diff;
    }
    return sum;
}

__global__ void findNearestCentroidBasicKernel(const float* points, const float* centroids, int* labels, int numPoints, int numCentroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        float minDistance = INFINITY;
        int nearestCentroid = 0;
        for (int c = 0; c < numCentroids; c++) {
            float distance = calculateDistanceBasic(&points[idx * dims], &centroids[c * dims], dims);
            if (distance < minDistance) {
                minDistance = distance;
                nearestCentroid = c;
            }
        }
        labels[idx] = nearestCentroid;
    }
}

__global__ void updateCentroidsBasicKernel(const float* points, const int* labels, float* newCentroids, int* clusterSizes, int numPoints, int numCentroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        int label = labels[idx];
        for (int d = 0; d < dims; d++) {
            atomicAdd(&newCentroids[label * dims + d], points[idx * dims + d]);
        }
        atomicAdd(&clusterSizes[label], 1);
    }
}

KMeansResult runCUDABasicKMeans(const std::vector<float>& points, const KMeansParams& params) {
    int numPoints = points.size() / params.dims;
    
    // Create device vectors
    thrust::device_vector<float> d_points = points;
    thrust::device_vector<float> d_centroids(params.k * params.dims);
    thrust::device_vector<float> d_newCentroids(params.k * params.dims);
    thrust::device_vector<int> d_labels(numPoints);
    thrust::device_vector<int> d_clusterSizes(params.k);
    thrust::device_vector<int> d_converged(1);

    // Initialize centroids
    int blockSize = 256;
    int gridSize = (params.k + blockSize - 1) / blockSize;
    initializeCentroidsKernel<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(d_centroids.data()),
        thrust::raw_pointer_cast(d_points.data()),
        numPoints, params.k, params.dims, params.seed
    );

    int iterations = 0;
    bool converged = false;
    while (iterations < params.maxIterations && !converged) {
        // Find nearest centroid for each point
        gridSize = (numPoints + blockSize - 1) / blockSize;
        findNearestCentroidBasicKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_points.data()),
            thrust::raw_pointer_cast(d_centroids.data()),
            thrust::raw_pointer_cast(d_labels.data()),
            numPoints, params.k, params.dims
        );

        // Reset new centroids and cluster sizes
        thrust::fill(d_newCentroids.begin(), d_newCentroids.end(), 0.0f);
        thrust::fill(d_clusterSizes.begin(), d_clusterSizes.end(), 0);

        // Update centroids
        updateCentroidsBasicKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_points.data()),
            thrust::raw_pointer_cast(d_labels.data()),
            thrust::raw_pointer_cast(d_newCentroids.data()),
            thrust::raw_pointer_cast(d_clusterSizes.data()),
            numPoints, params.k, params.dims
        );

        // Normalize centroids
        gridSize = (params.k + blockSize - 1) / blockSize;
        normalizeCentroidsKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_newCentroids.data()),
            thrust::raw_pointer_cast(d_clusterSizes.data()),
            params.k, params.dims
        );

        // Check for convergence
        thrust::fill(d_converged.begin(), d_converged.end(), 1);
        checkConvergenceKernel<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(d_centroids.data()),
            thrust::raw_pointer_cast(d_newCentroids.data()),
            params.k, params.dims, params.threshold,
            thrust::raw_pointer_cast(d_converged.data())
        );

        converged = d_converged[0] == 1;

        // Update centroids for next iteration
        d_centroids = d_newCentroids;

        iterations++;
    }

    // Copy results back to host
    thrust::host_vector<int> h_labels = d_labels;
    thrust::host_vector<float> h_centroids = d_centroids;

    std::vector<int> labels(h_labels.begin(), h_labels.end());
    std::vector<std::vector<float>> finalCentroids(params.k, std::vector<float>(params.dims));
    for (int i = 0; i < params.k; i++) {
        for (int d = 0; d < params.dims; d++) {
            finalCentroids[i][d] = h_centroids[i * params.dims + d];
        }
    }

    return KMeansResult{labels, finalCentroids, iterations, 0.0}; // Time per iteration is calculated in main.cpp
}