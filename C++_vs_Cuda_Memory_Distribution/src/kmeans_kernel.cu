#include "hip/hip_runtime.h"
// kmeans_kernel.cu
#include "kmeans_kernel.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__device__ float calculateDistanceDevice(const float* a, const float* b, int dims) {
    float sum = 0;
    for (int i = 0; i < dims; i++) {
        float diff = a[i] - b[i];
        sum += diff * diff;
    }
    return sum;
}

__global__ void initializeCentroidsKernel(float* centroids, const float* points, int numPoints, int numCentroids, int dims, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numCentroids) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        int randomIndex = hiprand(&state) % numPoints;
        for (int d = 0; d < dims; d++) {
            centroids[idx * dims + d] = points[randomIndex * dims + d];
        }
    }
}

__global__ void findNearestCentroidKernel(const float* points, const float* centroids, int* labels, int numPoints, int numCentroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        float minDistance = INFINITY;
        int nearestCentroid = 0;
        for (int c = 0; c < numCentroids; c++) {
            float distance = calculateDistanceDevice(&points[idx * dims], &centroids[c * dims], dims);
            if (distance < minDistance) {
                minDistance = distance;
                nearestCentroid = c;
            }
        }
        labels[idx] = nearestCentroid;
    }
}

__global__ void updateCentroidsKernel(const float* points, const int* labels, float* newCentroids, int* clusterSizes, int numPoints, int numCentroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        int label = labels[idx];
        for (int d = 0; d < dims; d++) {
            atomicAdd(&newCentroids[label * dims + d], points[idx * dims + d]);
        }
        atomicAdd(&clusterSizes[label], 1);
    }
}

__global__ void normalizeCentroidsKernel(float* centroids, const int* clusterSizes, int numCentroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numCentroids) {
        int size = clusterSizes[idx];
        if (size > 0) {
            for (int d = 0; d < dims; d++) {
                centroids[idx * dims + d] /= size;
            }
        }
    }
}

__global__ void checkConvergenceKernel(const float* oldCentroids, const float* newCentroids, int numCentroids, int dims, float threshold, int* converged) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numCentroids) {
        float distance = calculateDistanceDevice(&oldCentroids[idx * dims], &newCentroids[idx * dims], dims);
        if (distance > threshold * threshold) {
            *converged = 0;
        }
    }
}