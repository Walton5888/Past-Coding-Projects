#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/extrema.h>
#include <thrust/random.h>
#include <cmath>
#include <limits>

struct CalculateDistance {
    const float* points;
    const float* centroids;
    int dims;
    int k;
    int numPoints;

    CalculateDistance(const float* _points, const float* _centroids, int _dims, int _k, int _numPoints) 
        : points(_points), centroids(_centroids), dims(_dims), k(_k), numPoints(_numPoints) {}

    __device__
    int operator()(int idx) const {
        const float* point = points + idx * dims;
        float min_dist = INFINITY;
        int nearest_centroid = 0;

        for (int i = 0; i < k; ++i) {
            float dist = 0;
            for (int d = 0; d < dims; ++d) {
                float diff = point[d] - centroids[i * dims + d];
                dist += diff * diff;
            }
            if (dist < min_dist) {
                min_dist = dist;
                nearest_centroid = i;
            }
        }

        return nearest_centroid;
    }
};

KMeansResult runThrustKMeans(const std::vector<float>& points, const KMeansParams& params) {
    int numPoints = points.size() / params.dims;

    // Create device vectors
    thrust::device_vector<float> d_points = points;
    thrust::device_vector<float> d_centroids(params.k * params.dims);
    thrust::device_vector<int> d_labels(numPoints);
    thrust::device_vector<float> d_newCentroids(params.k * params.dims);
    thrust::device_vector<int> d_clusterSizes(params.k);

    // Initialize centroids randomly
    thrust::default_random_engine rng(params.seed);
    thrust::uniform_int_distribution<int> dist(0, numPoints - 1);
    for (int i = 0; i < params.k; ++i) {
        int idx = dist(rng);
        thrust::copy(d_points.begin() + idx * params.dims, 
                     d_points.begin() + (idx + 1) * params.dims, 
                     d_centroids.begin() + i * params.dims);
    }

    int iterations = 0;
    bool converged = false;

    while (iterations < params.maxIterations && !converged) {
        // Assign points to nearest centroid
        thrust::transform(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(numPoints),
            d_labels.begin(),
            CalculateDistance(thrust::raw_pointer_cast(d_points.data()),
                              thrust::raw_pointer_cast(d_centroids.data()),
                              params.dims, params.k, numPoints)
        );

        // Reset new centroids and cluster sizes
        thrust::fill(d_newCentroids.begin(), d_newCentroids.end(), 0.0f);
        thrust::fill(d_clusterSizes.begin(), d_clusterSizes.end(), 0);

        // Update centroids
        for (int i = 0; i < numPoints; ++i) {
            int label = d_labels[i];
            for (int d = 0; d < params.dims; ++d) {
                d_newCentroids[label * params.dims + d] += d_points[i * params.dims + d];
            }
            d_clusterSizes[label]++;
        }

        // Normalize centroids
        for (int i = 0; i < params.k; ++i) {
            if (d_clusterSizes[i] > 0) {
                for (int d = 0; d < params.dims; ++d) {
                    d_newCentroids[i * params.dims + d] /= d_clusterSizes[i];
                }
            }
        }

        // Check for convergence
        float max_movement = 0.0f;
        for (int i = 0; i < params.k * params.dims; ++i) {
            float diff = d_centroids[i] - d_newCentroids[i];
            max_movement = max(max_movement, diff * diff);
        }

        converged = (std::sqrt(max_movement) <= params.threshold);

        // Update centroids for next iteration
        d_centroids = d_newCentroids;

        iterations++;
    }

    // Copy results back to host
    std::vector<int> labels(numPoints);
    thrust::copy(d_labels.begin(), d_labels.end(), labels.begin());

    std::vector<std::vector<float>> finalCentroids(params.k, std::vector<float>(params.dims));
    thrust::host_vector<float> h_centroids = d_centroids;
    for (int i = 0; i < params.k; i++) {
        for (int d = 0; d < params.dims; d++) {
            finalCentroids[i][d] = h_centroids[i * params.dims + d];
        }
    }

    return KMeansResult{labels, finalCentroids, iterations, 0.0}; // Time per iteration is calculated in main.cpp
}

